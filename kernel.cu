#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace std;

__global__ void exhausted_kernel(int Ni,  int Nj, int Nk,  int C, float* results)
{
    // calculate the indices for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    // check if the current combination of Ti, Tj, and Tk is valid
    if (i < Ni && j < Nj && k < Nk && (i * k + j * k + i * j) <= C)
    {
        // calculate the result of the formula for the current combination of Ti, Tj, and Tk
        float result = (float)Nk * Nj * Nj / (1.0f / (float)j + 1.0f / (float)i + 1.0f / (float)k);

        // store the result in the results array
        results[i * Nj * Nk + j * Nk + k] = result;
    }
    else
    {
        // store a zero in the results array for invalid combinations
        results[i * Nj * Nk + j * Nk + k] = 0.0f;
    }
}

int main()
{
    int Ni = 16384;
    int Nj = 16384;
    int Nk = 16384;
    int C = 1000000;

    // define the results vector
    float* results = (float*)calloc(Ni * Nj * Nk, sizeof(float));
    for (int i = 0; i < Ni * Nj * Nk; i++)
    {
        results[i] = 0.0f;
    }

    // allocate memory on the GPU for the results array
    float* dev_results;
    hipMalloc((void**)&dev_results, Ni * Nj * Nk * sizeof(float));

    int num_threads = 256;  // number of threads per block
    int num_blocks = (Ni * Nj * Nk + num_threads - 1) / num_threads;  // number of blocks needed

    dim3 grid(num_blocks, 1, 1);
    dim3 block(num_threads, 1, 1);

    // transfer the results from the GPU to the host (CPU)
    hipMemcpy(dev_results, results, Ni * Nj * Nk * sizeof(float), hipMemcpyHostToDevice);

    // launch the kernel function on the GPU
    exhausted_kernel << <grid, block >> > (Ni, Nj, Nk, C, dev_results);

    // transfer the results from the GPU to the host (CPU)
    hipMemcpy(results, dev_results, Ni * Nj * Nk * sizeof(float), hipMemcpyDeviceToHost);


    double objective_value = 0;
    double best_objective_value = DBL_MAX;
    double best_Ti = 0;
    double best_Tj = 0;
    double best_Tk = 0;

    // print the results
    for (int i = 1; i < Ni; i++)
    {
        for (int j = 1; j < Nj; j++)
        {
            for (int k = 1; k < Nk; k++)
            {
                objective_value = results[i * Nj * Nk + j * Nk + k];
                if (objective_value < best_objective_value)
                {
                    best_objective_value = objective_value;
                    best_Ti = i;
                    best_Tj = j;
                    best_Tk = k;
                }
                // std::cout << "Result for Ti = " << i << ", Tj = " << j << ", Tk = " << k << ": " << results[i * Nj * Nk + j * Nk + k];
            }
        }
    }

    cout << "Best values:" << endl;
    cout << "Ti = " << best_Ti << endl;
    cout << "Tj = " << best_Tj << endl;
    cout << "Tk = " << best_Tk << endl;

    free(results);
    // free the GPU memory
    hipFree(dev_results);

    // free the host (CPU) memory
    return 0;
}